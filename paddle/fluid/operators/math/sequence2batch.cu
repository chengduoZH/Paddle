#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#define EIGEN_USE_GPU
#include "paddle/fluid/operators/math/sequence2batch.h"

namespace paddle {
namespace operators {
namespace math {

template <typename T, bool IsSrcIndex>
__global__ void CopyMatrixRowsKernel(const T* src, T* dst, const size_t* index,
                                     int64_t height, int64_t width) {
  int idx = threadIdx.x;
  int idy = threadIdx.y;
  int id = blockIdx.x + idy * gridDim.x;
  while (id < height) {
    int src_idx = IsSrcIndex ? index[id] : id;
    int dst_idx = IsSrcIndex ? id : index[id];
    const T* src_data = src + src_idx * width;
    T* dst_data = dst + dst_idx * width;
    for (int i = idx; i < width; i += blockDim.x) {
      dst_data[i] = src_data[i];
    }
    id += blockDim.y * gridDim.x;
  }
}

template <typename T>
class CopyMatrixRowsFunctor<platform::CUDADeviceContext, T> {
 public:
  void operator()(const platform::CUDADeviceContext& context,
                  const framework::Tensor& src,
                  framework::Vector<size_t> index_lod, framework::Tensor* dst,
                  bool is_src_index) {
    auto src_dims = src.dims();
    auto dst_dims = dst->dims();
    PADDLE_ENFORCE_EQ(src_dims.size(), 2,
                      "The src must be matrix with rank 2.");
    PADDLE_ENFORCE_EQ(dst_dims.size(), 2,
                      "The dst must be matrix with rank 2.");
    PADDLE_ENFORCE_EQ(src_dims[1], dst_dims[1],
                      "The width of src and dst must be same.");
    auto height = dst_dims[0];
    auto width = dst_dims[1];
    auto* src_data = src.data<T>();
    auto* dst_data = dst->data<T>();

    const int thread_rows = 8;
    const int thread_cols = 128;
    dim3 threads(thread_cols, thread_rows);

    int max_threads = context.GetMaxPhysicalThreadCount();
    int max_blocks = std::max(max_threads / (thread_cols * thread_rows), 1);

    int grid_rows = std::min(
        max_blocks, std::max(static_cast<int>(height / thread_rows), 1));

    dim3 grid(grid_rows, 1);
    auto stream = context.stream();
    if (is_src_index) {
      CopyMatrixRowsKernel<T, true><<<grid, threads, 0, stream>>>(
          src_data, dst_data, index_lod.CUDAData(context.GetPlace()), height,
          width);
    } else {
      CopyMatrixRowsKernel<T, false><<<grid, threads, 0, stream>>>(
          src_data, dst_data, index_lod.CUDAData(context.GetPlace()), height,
          width);
    }
  }
};

template class CopyMatrixRowsFunctor<platform::CUDADeviceContext, float>;
template class CopyMatrixRowsFunctor<platform::CUDADeviceContext, double>;

template class LoDTensor2BatchFunctor<platform::CUDADeviceContext, float>;
template class LoDTensor2BatchFunctor<platform::CUDADeviceContext, double>;
template class Batch2LoDTensorFunctor<platform::CUDADeviceContext, float>;
template class Batch2LoDTensorFunctor<platform::CUDADeviceContext, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
