#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/data_device_transform.h"

namespace paddle {
namespace framework {

extern proto::VarType::Type ToDataType(std::type_index type);

template <typename T>
__global__ void KeDataTransFromCPUToGPU(const T in_value, T* out_value) {
  out_value[0] = in_value;
}

void DataTransFromCPUToGPU(const Tensor& src, const platform::Place& dst_place,
                           const platform::DeviceContext& dev_ctx,
                           Tensor* dst) {
  src.check_memory_size();
  auto type = src.type();

  dst->Resize(src.dims());
  dst->set_layout(src.layout());
  dst->mutable_data(dst_place, type);

  auto& cuda_ctx =
      reinterpret_cast<const platform::CUDADeviceContext&>(dev_ctx);

  switch (ToDataType(type)) {
    case proto::VarType::FP32:
      KeDataTransFromCPUToGPU<<<1, 1, 0, cuda_ctx.stream()>>><float>(
          src.data<float>()[0], dst->mutable_data<float>(dst_place));
      break;
    case proto::VarType::FP64:
      KeDataTransFromCPUToGPU<<<1, 1, 0, cuda_ctx.stream()>>><double>(
          src.data<double>()[0], dst->mutable_data<double>(dst_place));
      break;
    case proto::VarType::INT32:
      KeDataTransFromCPUToGPU<<<1, 1, 0, cuda_ctx.stream()>>><int>(
          src.data<int>()[0], dst->mutable_data<int>(dst_place));
      break;
    case proto::VarType::INT64:
      KeDataTransFromCPUToGPU<<<1, 1, 0, cuda_ctx.stream()>>><int64_t>(
          src.data<int64_t>()[0], dst->mutable_data<int64_t>(dst_place));
      break;
    default:
      PADDLE_THROW("Not supported %d", type);
  }
}

static const platform::DeviceContext* GetDeviceContext(
    const platform::Place& src_place, const platform::Place& dst_place) {
  platform::DeviceContextPool& pool = platform::DeviceContextPool::Instance();

  if (platform::is_gpu_place(src_place) && platform::is_cpu_place(dst_place)) {
    return pool.Get(src_place);
  } else if (platform::is_cpu_place(src_place) &&
             platform::is_gpu_place(dst_place)) {
    return pool.Get(dst_place);
  } else {
    PADDLE_THROW(
        "Currently, model parallelism is only supported between CPU and CUDA");
  }
}

void TransDataDevice(const Tensor& in, const platform::Place& dst_place,
                     Tensor* out) {
  VLOG(3) << "DeviceTransform in, src_place " << in.place()
          << " dst_place: " << dst_place << "  " << in.dims();
  auto* dev_ctx = GetDeviceContext(in.place(), dst_place);

  // FIXME(zcd): TransDataDevice is used to transform data from GPU to CPU and
  // the enforced checkings have been done in GetDeviceContext, so the
  // `dev_ctx->Wait()` is necessary. But `dev_ctx->Wait()` will make the program
  // slow, especially the number of elements is one. So one solution is to use a
  // cuda kernel to complete the copy operation when the transforming is from
  // CPU to GPU and the number of elements is one.
  if (platform::is_cpu_place(in.place()) && platform::is_gpu_place(dst_place) &&
      in.numel() == 1) {
    PADDLE_ENFORCE(platform::is_gpu_place(dev_ctx->GetPlace()));
#ifdef __NVCC__
    DataTransFromCPUToGPU(in, dst_place, *dev_ctx, out);
#endif
  } else {
    TensorCopy(in, dst_place, *dev_ctx, out);
    dev_ctx->Wait();
  }
}

}  // namespace framework
}  // namespace paddle
