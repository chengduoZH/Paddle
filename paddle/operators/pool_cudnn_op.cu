#include "hip/hip_runtime.h"
/* Copyright (c) 2016 PaddlePaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/pool_cudnn_op.h"
#include "paddle/platform/cudnn_helper.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using ScopedTensorDescriptor = platform::ScopedTensorDescriptor;
using ScopedPoolingDescriptor = platform::ScopedPoolingDescriptor;
using DataLayout = platform::DataLayout;
using PoolingMode = platform::PoolingMode;

// NOTE: copy from conv_cudnn
std::vector<int> Dims2Vector(const framework::DDim &dims) {
  std::vector<int> ret;
  for (int i = 0; i < dims.size(); i++) {
    ret.push_back(dims[i]);
  }
  return ret;
}

template <typename T>
class PoolCudnnOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");

    const Tensor *input = ctx.Input<Tensor>("X");
    Tensor *output = ctx.Output<Tensor>("Out");

    std::string pooling_type = ctx.Attr<std::string>("poolingType");
    std::vector<int> ksize = ctx.Attr<std::vector<int>>("ksize");
    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");
    if (ctx.Attr<bool>("globalPooling")) {
      for (size_t i = 0; i < ksize.size(); ++i) {
        ksize[i] = static_cast<int>(input->dims()[i + 2]);
      }
    }

    const T *input_data = input->data<T>();
    T *output_data = output->mutable_data<T>(ctx.GetPlace());

    // ------------------- cudnn descriptors ---------------------
    ScopedTensorDescriptor input_desc;
    ScopedTensorDescriptor output_desc;
    ScopedPoolingDescriptor pool_desc;

    // ------------------- cudnn pool algorithm ---------------------
    auto handle = ctx.cuda_device_context().cudnn_handle();
    T alpha = 1.0f, beta = 1.0f;

    /*
    PoolingMode pooling_mode;
    DataLayout layout;
    // In the infershape phase, pooling_type has been limited in {"max" or
    // "avg"}
    if (pooling_type == "max") {
      pooling_mode = PoolingMode::kMaximum;
    } else {
      pooling_mode = PoolingMode::kAverage;
    }
    if (ksize.size() == 2) {
      layout = DataLayout::kNCHW;
    } else {
      // TODO(chengduo)
    }
    */
    hipdnnTensorDescriptor_t cudnn_input_desc =
        input_desc.descriptor<T>(DataLayout::kNCHW, Dims2Vector(input->dims()));
    hipdnnTensorDescriptor_t cudnn_output_desc = output_desc.descriptor<T>(
        DataLayout::kNCHW, Dims2Vector(output->dims()));

    hipdnnPoolingDescriptor_t cudnn_pool_desc =
        pool_desc.descriptor(PoolingMode::kMaximum, ksize, paddings, strides);

    PADDLE_ENFORCE(platform::dynload::hipdnnPoolingForward(
        handle, cudnn_pool_desc, &alpha, cudnn_input_desc, input_data, &beta,
        cudnn_output_desc, output_data));

    switch (ksize.size()) {
      case 2: {
      } break;
      case 3: {
      } break;
    }
  }
};

template <typename T>
class PoolCudnnGradOpKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "It must use GPUPlace.");

    const Tensor *input = ctx.Input<Tensor>("X");
    const Tensor *output = ctx.Input<Tensor>("Out");
    const Tensor *output_grad =
        ctx.Input<Tensor>(framework::GradVarName("Out"));
    Tensor *input_grad = ctx.Output<Tensor>(framework::GradVarName("X"));

    std::string pooling_type = ctx.Attr<std::string>("poolingType");
    std::vector<int> ksize = ctx.Attr<std::vector<int>>("ksize");
    std::vector<int> strides = ctx.Attr<std::vector<int>>("strides");
    std::vector<int> paddings = ctx.Attr<std::vector<int>>("paddings");

    if (ctx.Attr<bool>("globalPooling")) {
      for (size_t i = 0; i < ksize.size(); ++i)
        ksize[i] = static_cast<int>(input->dims()[i + 2]);
    }

    const T *input_data = input->data<T>();
    const T *output_data = output->data<T>();
    const T *output_grad_data = output_grad->data<T>();
    T *input_grad_data = input_grad->mutable_data<T>(ctx.GetPlace());

    // ------------------- cudnn descriptors ---------------------
    ScopedTensorDescriptor input_desc;
    ScopedTensorDescriptor output_desc;
    ScopedTensorDescriptor output_grad_desc;
    ScopedTensorDescriptor input_grad_desc;
    ScopedPoolingDescriptor pool_desc;

    // ------------------- cudnn backward algorithm ---------------------
    auto handle = ctx.cuda_device_context().cudnn_handle();
    T alpha = 1.0f, beta = 1.0f;

    if (input_grad) {
      input_grad->mutable_data<T>(ctx.GetPlace());
      auto temp = framework::EigenVector<T>::Flatten(*input_grad);
      temp.device(ctx.GetEigenDevice<paddle::platform::GPUPlace>()) =
          temp.constant(static_cast<T>(0));

      /*
      PoolingMode pooling_mode;
      DataLayout layout;
      // In the infershape phase, pooling_type has been limited in {"max" or
      // "avg"}
      if (pooling_type == "max") {
        pooling_mode = PoolingMode::kMaximum;
      } else {
        pooling_mode = PoolingMode::kAverage;
      }
      if (ksize.size() == 2) {
        layout = DataLayout::kNCHW;
      } else {
        // TODO(chengduo)
      }
      */
      hipdnnTensorDescriptor_t cudnn_input_desc = input_desc.descriptor<T>(
          DataLayout::kNCHW, Dims2Vector(input->dims()));
      hipdnnTensorDescriptor_t cudnn_output_desc = input_desc.descriptor<T>(
          DataLayout::kNCHW, Dims2Vector(output->dims()));
      hipdnnTensorDescriptor_t cudnn_input_grad_desc =
          input_grad_desc.descriptor<T>(DataLayout::kNCHW,
                                        Dims2Vector(input_grad->dims()));
      hipdnnTensorDescriptor_t cudnn_output_grad_desc =
          output_grad_desc.descriptor<T>(DataLayout::kNCHW,
                                         Dims2Vector(output_grad->dims()));

      hipdnnPoolingDescriptor_t cudnn_pool_desc =
          pool_desc.descriptor(PoolingMode::kMaximum, ksize, paddings, strides);

      PADDLE_ENFORCE(platform::dynload::hipdnnPoolingBackward(
          handle, cudnn_pool_desc, &alpha, cudnn_output_desc, output_data,
          cudnn_output_grad_desc, output_grad_data, cudnn_input_desc,
          input_data, &beta, cudnn_input_grad_desc, input_grad_data));

      switch (ksize.size()) {
        case 2: {
        } break;
        case 3: {
        } break;
      }
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;

REGISTER_OP_GPU_KERNEL(pool2d_cudnn, ops::PoolCudnnOpKernel<float>);
REGISTER_OP_GPU_KERNEL(pool2d_cudnn_grad, ops::PoolCudnnGradOpKernel<float>);

REGISTER_OP_GPU_KERNEL(pool3d_cudnn, ops::PoolCudnnOpKernel<float>);
REGISTER_OP_GPU_KERNEL(pool3d_cudnn_grad, ops::PoolCudnnGradOpKernel<float>);
