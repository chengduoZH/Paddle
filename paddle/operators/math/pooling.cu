#include "hip/hip_runtime.h"
/* Copyright (c) 2016 paddlepaddle Authors. All Rights Reserve.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/operators/math/pooling.h"
#include "paddle/platform/cuda_helper.h"

namespace paddle {
namespace operators {
namespace math {

template <typename PoolProcess, typename T>
__global__ void KernelPool2dForward(
    const int nthreads, const T* input_data, T* output_data, const int channels,
    const int input_height, const int input_width, const int output_height,
    const int output_width, const int ksize_height, const int ksize_width,
    const int stride_height, const int stride_width, const int padding_height,
    const int padding_width, PoolProcess pool_compute) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart = ph * stride_height - padding_height;
    int hend = min(hstart + ksize_height, input_height);
    hstart = max(hstart, 0);

    int wstart = pw * stride_width - padding_width;
    int wend = min(wstart + ksize_width, input_width);
    wstart = max(wstart, 0);

    input_data += (batch_idx * channels + c) * input_height * input_width;
    T ele = pool_compute.initial();
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        pool_compute.compute(ele, input_data[h * input_width + w]);
      }
    }
    int pool_size = (hend - hstart) * (wend - wstart);
    pool_compute.finalize(ele, (static_cast<T>(pool_size)));
    output_data[index] = ele;
  }
}

template <typename PoolProcess, typename T>
__global__ void KernelPool2dBackward(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, T* input_grad, const int channels,
    const int input_height, const int input_width, const int output_height,
    const int output_width, const int ksize_height, const int ksize_width,
    const int stride_height, const int stride_width, const int padding_height,
    const int padding_width, PoolProcess pool_compute) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    int offsetW = index % input_width + padding_width;
    int offsetH = (index / input_width) % input_height + padding_height;
    int offsetC = (index / input_width / input_height) % channels;
    int batch_idx = index / input_width / input_height / channels;

    int phstart = (offsetH < ksize_height)
                      ? 0
                      : (offsetH - ksize_height) / stride_height + 1;
    int pwstart = (offsetW < ksize_width)
                      ? 0
                      : (offsetW - ksize_width) / stride_width + 1;
    int phend = min(offsetH / stride_height + 1, output_height);
    int pwend = min(offsetW / stride_width + 1, output_width);
    T gradient = 0;
    T input = input_data[index];
    int output_idx =
        (batch_idx * channels + offsetC) * output_height * output_width;
    output_data += output_idx;
    output_grad += output_idx;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        int hstart = ph * stride_height - padding_height;
        int wstart = pw * stride_width - padding_width;
        int hend = min(hstart + ksize_height, input_height);
        int wend = min(wstart + ksize_width, input_width);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
        int pool_size = (hend - hstart) * (wend - wstart);
        int output_sub_idx = ph * output_width + pw;
        pool_compute.compute(input, output_data[output_sub_idx],
                             output_grad[output_sub_idx], gradient,
                             static_cast<T>(1.0 / pool_size));
      }
    }
    input_grad[index] = gradient;
  }
}

template <typename T>
__global__ void KernelMaxPool2dBackward(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, T* input_grad, const int channels,
    const int input_height, const int input_width, const int output_height,
    const int output_width, const int ksize_height, const int ksize_width,
    const int stride_height, const int stride_width, const int padding_height,
    const int padding_width) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int c = (index / output_width / output_height) % channels;
    int batch_idx = index / output_width / output_height / channels;

    int hstart = ph * stride_height - padding_height;
    int hend = min(hstart + ksize_height, input_height);
    hstart = max(hstart, 0);

    int wstart = pw * stride_width - padding_width;
    int wend = min(wstart + ksize_width, input_width);
    wstart = max(wstart, 0);

    input_data += (batch_idx * channels + c) * input_height * input_width;
    input_grad += (batch_idx * channels + c) * input_height * input_width;

    T ele = output_data[index];
    int maxIndex = -1;
    bool stop = false;
    for (int h = hstart; h < hend && !stop; ++h) {
      for (int w = wstart; w < wend && !stop; ++w) {
        if (ele == input_data[h * input_width + w]) {
          maxIndex = h * input_width + w;
          stop = true;
        }
      }
    }

    if (maxIndex != -1) {
      // atomic add
      atomicAdd(input_grad + maxIndex, output_grad[index]);
    }
  }
}

template <typename PoolProcess, typename T>
class Pool2dFunctor<platform::GPUPlace, PoolProcess, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& output,
                  std::vector<int>& ksize, std::vector<int>& strides,
                  std::vector<int>& paddings, PoolProcess pool_compute) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output.dims()[1];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    T* output_data = output.mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2dForward<
        PoolProcess,
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(nthreads, input_data, output_data, input_channels,
                              input_height, input_width, output_height,
                              output_width, ksize_height, ksize_width,
                              stride_height, stride_width, padding_height,
                              padding_width, pool_compute);
  }
};

template <typename PoolProcess, typename T>
class Pool2dGradFunctor<platform::GPUPlace, PoolProcess, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& input_grad,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad, std::vector<int>& ksize,
                  std::vector<int>& strides, std::vector<int>& paddings,
                  PoolProcess pool_compute) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad.mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * input_channels * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool2dBackward<
        PoolProcess,
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_grad_data,
        input_channels, input_height, input_width, output_height, output_width,
        ksize_height, ksize_width, stride_height, stride_width, padding_height,
        padding_width, pool_compute);
  }
};

template <typename T>
class MaxPool2dGradFunctor<platform::GPUPlace, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& input_grad,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad, std::vector<int>& ksize,
                  std::vector<int>& strides, std::vector<int>& paddings) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_height = input.dims()[2];
    const int input_width = input.dims()[3];
    const int output_channels = output.dims()[1];
    const int output_height = output.dims()[2];
    const int output_width = output.dims()[3];
    const int ksize_height = ksize[0];
    const int ksize_width = ksize[1];
    const int stride_height = strides[0];
    const int stride_width = strides[1];
    const int padding_height = paddings[0];
    const int padding_width = paddings[1];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad.mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_height * output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool2dBackward<
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_grad_data,
        input_channels, input_height, input_width, output_height, output_width,
        ksize_height, ksize_width, stride_height, stride_width, padding_height,
        padding_width);
  }
};

template class MaxPool2dGradFunctor<platform::GPUPlace, float>;
// template class MaxPool2dGradFunctor<platform::GPUPlace, double>;

template class Pool2dFunctor<platform::GPUPlace,
                             paddle::operators::math::maxPool<float>, float>;
template class Pool2dFunctor<platform::GPUPlace,
                             paddle::operators::math::avgPool<float>, float>;
template class Pool2dGradFunctor<
    platform::GPUPlace, paddle::operators::math::maxPoolGrad<float>, float>;
template class Pool2dGradFunctor<
    platform::GPUPlace, paddle::operators::math::avgPoolGrad<float>, float>;
template class Pool2dFunctor<platform::GPUPlace,
                             paddle::operators::math::maxPool<double>, double>;
template class Pool2dFunctor<platform::GPUPlace,
                             paddle::operators::math::avgPool<double>, double>;
template class Pool2dGradFunctor<
    platform::GPUPlace, paddle::operators::math::maxPoolGrad<double>, double>;
template class Pool2dGradFunctor<
    platform::GPUPlace, paddle::operators::math::avgPoolGrad<double>, double>;

template <typename PoolProcess, typename T>
__global__ void KernelPool3DForward(
    const int nthreads, const T* input_data, T* output_data, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    PoolProcess pool_compute) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < (nthreads);
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;
    int dstart = pd * stride_depth - padding_depth;
    int hstart = ph * stride_height - padding_height;
    int wstart = pw * stride_width - padding_width;
    int dend = min(dstart + ksize_depth, input_depth);
    int hend = min(hstart + ksize_height, input_height);
    int wend = min(wstart + ksize_width, input_width);
    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T ele = pool_compute.initial();
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;
    for (int d = dstart; d < dend; ++d) {
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          pool_compute.compute(
              ele, input_data[(d * input_height + h) * input_width + w]);
        }
      }
    }
    int pool_size = (dend - dstart) * (hend - hstart) * (wend - wstart);
    pool_compute.finalize(ele, static_cast<T>(pool_size));
    output_data[index] = ele;
  }
}

template <typename PoolProcess, typename T>
__global__ void KernelPool3DBackward(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, T* input_grad, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height, const int padding_width,
    PoolProcess pool_compute) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < (nthreads);
       index += blockDim.x * gridDim.x) {
    int offsetW = index % input_width + padding_width;
    int offsetH = (index / input_width) % input_height + padding_height;
    int offsetD =
        (index / input_width / input_height) % input_depth + padding_depth;
    int offsetC = (index / input_width / input_height / input_depth) % channels;
    int batch_idx = index / input_width / input_height / input_depth / channels;

    int pdstart = (offsetD < ksize_depth)
                      ? 0
                      : (offsetD - ksize_depth) / stride_depth + 1;
    int phstart = (offsetH < ksize_height)
                      ? 0
                      : (offsetH - ksize_height) / stride_height + 1;
    int pwstart = (offsetW < ksize_width)
                      ? 0
                      : (offsetW - ksize_width) / stride_width + 1;
    int pdend = min((offsetD) / stride_depth + 1, output_depth);
    int phend = min((offsetH) / stride_height + 1, output_height);
    int pwend = min((offsetW) / stride_width + 1, output_width);

    T gradient = 0;
    T input = input_data[index];
    int output_idx = (batch_idx * channels + offsetC) * output_depth *
                     output_height * output_width;
    output_data += output_idx;
    output_grad += output_idx;

    for (int pd = pdstart; pd < pdend; ++pd) {
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          // figure out the pooling size
          int dstart = pd * stride_depth - padding_depth;
          int hstart = ph * stride_height - padding_height;
          int wstart = pw * stride_width - padding_width;
          int dend = min(dstart + ksize_depth, input_depth);
          int hend = min(hstart + ksize_height, input_height);
          int wend = min(wstart + ksize_width, input_width);
          dstart = max(dstart, 0);
          hstart = max(hstart, 0);
          wstart = max(wstart, 0);
          int pool_size = (dend - dstart) * (hend - hstart) * (wend - wstart);
          int output_sub_idx = (pd * output_height + ph) * output_width + pw;
          pool_compute.compute(input, output_data[output_sub_idx],
                               output_grad[output_sub_idx], gradient,
                               static_cast<T>(1.0 / pool_size));
        }
      }
    }
    input_grad[index] = gradient;
  }
}

template <typename T>
__global__ void KernelMaxPool3DBackward(
    const int nthreads, const T* input_data, const T* output_data,
    const T* output_grad, T* input_grad, const int channels,
    const int input_depth, const int input_height, const int input_width,
    const int output_depth, const int output_height, const int output_width,
    const int ksize_depth, const int ksize_height, const int ksize_width,
    const int stride_depth, const int stride_height, const int stride_width,
    const int padding_depth, const int padding_height,
    const int padding_width) {
  for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < (nthreads);
       index += blockDim.x * gridDim.x) {
    int pw = index % output_width;
    int ph = (index / output_width) % output_height;
    int pd = (index / output_width / output_height) % output_depth;
    int c = (index / output_width / output_height / output_depth) % channels;
    int batch_idx =
        index / output_width / output_height / output_depth / channels;
    int dstart = pd * stride_depth - padding_depth;
    int hstart = ph * stride_height - padding_height;
    int wstart = pw * stride_width - padding_width;
    int dend = min(dstart + ksize_depth, input_depth);
    int hend = min(hstart + ksize_height, input_height);
    int wend = min(wstart + ksize_width, input_width);
    dstart = max(dstart, 0);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    T ele = output_data[index];
    bool stop = false;
    int maxIdx = -1;
    input_data +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;
    input_grad +=
        (batch_idx * channels + c) * input_depth * input_height * input_width;

    for (int d = dstart; d < dend && !stop; ++d) {
      for (int h = hstart; h < hend && !stop; ++h) {
        for (int w = wstart; w < wend && !stop; ++w) {
          if (ele == input_data[(d * input_height + h) * input_width + w]) {
            stop = true;
            maxIdx = (d * input_height + h) * input_width + w;
          }
        }
      }
    }
    if (maxIdx != -1) {
      // atomic add
      atomicAdd(input_grad + maxIdx, output_grad[index]);
    }
  }
}

template <typename PoolProcess, class T>
class Pool3dFunctor<platform::GPUPlace, PoolProcess, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& output,
                  std::vector<int>& ksize, std::vector<int>& strides,
                  std::vector<int>& paddings, PoolProcess pool_compute) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    T* output_data = output.mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3DForward<
        PoolProcess,
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(
        nthreads, input_data, output_data, input_channels, input_depth,
        input_height, input_width, output_depth, output_height, output_width,
        ksize_depth, ksize_height, ksize_width, stride_depth, stride_height,
        stride_width, padding_depth, padding_height, padding_width,
        pool_compute);
  }
};

template <typename PoolProcess, class T>
class Pool3dGradFunctor<platform::GPUPlace, PoolProcess, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& input_grad,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad, std::vector<int>& ksize,
                  std::vector<int>& strides, std::vector<int>& paddings,
                  PoolProcess pool_compute) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad.mutable_data<T>(context.GetPlace());

    int nthreads =
        batch_size * input_channels * input_depth * input_height * input_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelPool3DBackward<
        PoolProcess,
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_grad_data,
        input_channels, input_depth, input_height, input_width, output_depth,
        output_height, output_width, ksize_depth, ksize_height, ksize_width,
        stride_depth, stride_height, stride_width, padding_depth,
        padding_height, padding_width, pool_compute);
  }
};

template <class T>
class MaxPool3dGradFunctor<platform::GPUPlace, T> {
 public:
  void operator()(const platform::DeviceContext& context,
                  const framework::Tensor& input, framework::Tensor& input_grad,
                  const framework::Tensor& output,
                  const framework::Tensor& output_grad, std::vector<int>& ksize,
                  std::vector<int>& strides, std::vector<int>& paddings) {
    const int batch_size = input.dims()[0];
    const int input_channels = input.dims()[1];
    const int input_depth = input.dims()[2];
    const int input_height = input.dims()[3];
    const int input_width = input.dims()[4];
    const int output_channels = output.dims()[1];
    const int output_depth = output.dims()[2];
    const int output_height = output.dims()[3];
    const int output_width = output.dims()[4];
    const int ksize_depth = ksize[0];
    const int ksize_height = ksize[1];
    const int ksize_width = ksize[2];
    const int stride_depth = strides[0];
    const int stride_height = strides[1];
    const int stride_width = strides[2];
    const int padding_depth = paddings[0];
    const int padding_height = paddings[1];
    const int padding_width = paddings[2];

    const T* input_data = input.data<T>();
    const T* output_data = output.data<T>();
    const T* output_grad_data = output_grad.data<T>();
    T* input_grad_data = input_grad.mutable_data<T>(context.GetPlace());

    int nthreads = batch_size * output_channels * output_depth * output_height *
                   output_width;
    int blocks = (nthreads + 1024 - 1) / 1024;
    dim3 threads(1024, 1);
    dim3 grid(blocks, 1);

    KernelMaxPool3DBackward<
        T><<<grid, threads, 0,
             reinterpret_cast<const platform::CUDADeviceContext&>(context)
                 .stream()>>>(
        nthreads, input_data, output_data, output_grad_data, input_grad_data,
        input_channels, input_depth, input_height, input_width, output_depth,
        output_height, output_width, ksize_depth, ksize_height, ksize_width,
        stride_depth, stride_height, stride_width, padding_depth,
        padding_height, padding_width);
  }
};

template class MaxPool3dGradFunctor<platform::GPUPlace, float>;
// template class MaxPool3dGradFunctor<platform::GPUPlace, double>;

template class Pool3dFunctor<platform::GPUPlace,
                             paddle::operators::math::maxPool<float>, float>;
template class Pool3dFunctor<platform::GPUPlace,
                             paddle::operators::math::avgPool<float>, float>;
template class Pool3dGradFunctor<
    platform::GPUPlace, paddle::operators::math::maxPoolGrad<float>, float>;
template class Pool3dGradFunctor<
    platform::GPUPlace, paddle::operators::math::avgPoolGrad<float>, float>;
template class Pool3dFunctor<platform::GPUPlace,
                             paddle::operators::math::maxPool<double>, double>;
template class Pool3dFunctor<platform::GPUPlace,
                             paddle::operators::math::avgPool<double>, double>;
template class Pool3dGradFunctor<
    platform::GPUPlace, paddle::operators::math::maxPoolGrad<double>, double>;
template class Pool3dGradFunctor<
    platform::GPUPlace, paddle::operators::math::avgPoolGrad<double>, double>;

}  // namespace math
}  // namespace operators
}  // namespace paddle
